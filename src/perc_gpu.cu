#include "hip/hip_runtime.h"
// -*- mode: C++; -*-
//
// Copyright (C) 2025, Rupert Nash, The University of Edinburgh.
//
// All rights reserved.
//
// This file is provided to you to complete an assessment and for
// subsequent private study. It may not be shared and, in particular,
// may not be posted on the internet. Sharing this or any modified
// version may constitute academic misconduct under the University's
// regulations.

#include "perc_gpu.h"

#include <cstdio>
#include <cstring>
#include <vector>
#include <hip/hip_runtime.h>

// Configuration constants
constexpr int BLOCK_SIZE = 16;  // Back to 16 for safer memory access
constexpr int printfreq = 100;

// CUDA error checking macro
#define CHECK_CUDA_ERROR(val) check_cuda( (val), #val, __FILE__, __LINE__ )
template<typename T>
void check_cuda(T result, char const *const func, const char *const file, int const line) {
    if (result) {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \n",
                file, line, static_cast<unsigned int>(result), func);
        exit(EXIT_FAILURE);
    }
}

// Optimized CUDA kernel using shared memory
__global__ void percolate_kernel(int M, int N, const int* __restrict__ state, 
                                int* __restrict__ next, int* changes) {
    extern __shared__ int shared_state[];
    
    // Calculate global indices
    const int i = blockIdx.y * blockDim.y + threadIdx.y;
    const int j = blockIdx.x * blockDim.x + threadIdx.x;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    
    // Shared memory index
    const int smem_idx = ty * blockDim.x + tx;
    
    // Load data into shared memory
    if (i <= M && j <= N) {
        const int global_idx = i * (N + 2) + j;
        shared_state[smem_idx] = state[global_idx];
    }
    
    __syncthreads();
    
    // Process only interior points
    if (i >= 1 && i <= M && j >= 1 && j <= N) {
        const int oldval = shared_state[smem_idx];
        int newval = oldval;
        
        if (oldval != 0) {
            // Check neighbors from global memory (safer but slower)
            const int global_idx = i * (N + 2) + j;
            newval = max(newval, state[global_idx - (N + 2)]);  // Up
            newval = max(newval, state[global_idx + (N + 2)]);  // Down
            newval = max(newval, state[global_idx - 1]);        // Left
            newval = max(newval, state[global_idx + 1]);        // Right
            
            next[global_idx] = newval;
            
            if (newval != oldval) {
                atomicAdd(changes, 1);
            }
        } else {
            next[i * (N + 2) + j] = 0;
        }
    }
}

struct GpuRunner::Impl {
    int M;
    int N;
    int* state;      // Host memory (pinned)
    int* tmp;        // Host memory (pinned)
    int* d_state;    // Device memory
    int* d_tmp;      // Device memory
    int* d_changes;  // Device memory for counting changes
    int* h_changes;  // Host memory for changes (pinned)
    
    // CUDA events for timing
    hipEvent_t start_event;
    hipEvent_t stop_event;
    hipEvent_t transfer_start;
    hipEvent_t transfer_stop;
    
    float kernel_ms;    // Kernel execution time
    float transfer_ms;  // Transfer time

    Impl(int m, int n) : M(m), N(n), kernel_ms(0), transfer_ms(0) {
        // Allocate pinned memory for better transfer speeds
        CHECK_CUDA_ERROR(hipHostAlloc(&state, size() * sizeof(int), hipHostMallocDefault));
        CHECK_CUDA_ERROR(hipHostAlloc(&tmp, size() * sizeof(int), hipHostMallocDefault));
        CHECK_CUDA_ERROR(hipHostAlloc(&h_changes, sizeof(int), hipHostMallocDefault));
        
        // Allocate device memory
        CHECK_CUDA_ERROR(hipMalloc(&d_state, size() * sizeof(int)));
        CHECK_CUDA_ERROR(hipMalloc(&d_tmp, size() * sizeof(int)));
        CHECK_CUDA_ERROR(hipMalloc(&d_changes, sizeof(int)));
        
        // Create timing events
        CHECK_CUDA_ERROR(hipEventCreate(&start_event));
        CHECK_CUDA_ERROR(hipEventCreate(&stop_event));
        CHECK_CUDA_ERROR(hipEventCreate(&transfer_start));
        CHECK_CUDA_ERROR(hipEventCreate(&transfer_stop));
    }
    
    ~Impl() {
        // Free pinned memory
        CHECK_CUDA_ERROR(hipHostFree(state));
        CHECK_CUDA_ERROR(hipHostFree(tmp));
        CHECK_CUDA_ERROR(hipHostFree(h_changes));
        
        // Free device memory
        hipFree(d_state);
        hipFree(d_tmp);
        hipFree(d_changes);
        
        // Destroy events
        hipEventDestroy(start_event);
        hipEventDestroy(stop_event);
        hipEventDestroy(transfer_start);
        hipEventDestroy(transfer_stop);
    }

    int size() const {
        return (M + 2) * (N + 2);
    }
};

GpuRunner::GpuRunner(int M, int N) : m_impl(std::make_unique<Impl>(M, N)) {
}

GpuRunner::~GpuRunner() = default;

void GpuRunner::copy_in(int const* source) {
    // Record transfer start time
    CHECK_CUDA_ERROR(hipEventRecord(m_impl->transfer_start));
    
    // Copy to pinned memory then to device
    std::copy(source, source + m_impl->size(), m_impl->state);
    CHECK_CUDA_ERROR(hipMemcpy(m_impl->d_state, m_impl->state,
                               m_impl->size() * sizeof(int),
                               hipMemcpyHostToDevice));
    
    // Record transfer end time
    CHECK_CUDA_ERROR(hipEventRecord(m_impl->transfer_stop));
    CHECK_CUDA_ERROR(hipEventSynchronize(m_impl->transfer_stop));
    CHECK_CUDA_ERROR(hipEventElapsedTime(&m_impl->transfer_ms,
                                         m_impl->transfer_start,
                                         m_impl->transfer_stop));
}

void GpuRunner::copy_out(int* dest) const {
    // Record transfer start time
    CHECK_CUDA_ERROR(hipEventRecord(m_impl->transfer_start));
    
    // Copy from device to pinned memory then to destination
    CHECK_CUDA_ERROR(hipMemcpy(m_impl->state, m_impl->d_state,
                               m_impl->size() * sizeof(int),
                               hipMemcpyDeviceToHost));
    std::copy(m_impl->state, m_impl->state + m_impl->size(), dest);
    
    // Record transfer end time
    CHECK_CUDA_ERROR(hipEventRecord(m_impl->transfer_stop));
    CHECK_CUDA_ERROR(hipEventSynchronize(m_impl->transfer_stop));
    float transfer_ms;
    CHECK_CUDA_ERROR(hipEventElapsedTime(&transfer_ms,
                                         m_impl->transfer_start,
                                         m_impl->transfer_stop));
    m_impl->transfer_ms += transfer_ms;
}

void GpuRunner::run() {
    int const M = m_impl->M;
    int const N = m_impl->N;
    
    // Calculate grid and block dimensions
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((N + BLOCK_SIZE - 1) / BLOCK_SIZE,
                 (M + BLOCK_SIZE - 1) / BLOCK_SIZE);
    
    // Calculate shared memory size
    size_t smem_size = BLOCK_SIZE * BLOCK_SIZE * sizeof(int);

    int const maxstep = 4 * std::max(M, N);
    int step = 1;
    int nchange = 1;

    // Use pointers to device buffers
    int* d_current = m_impl->d_state;
    int* d_next = m_impl->d_tmp;
    
    // Record kernel start time
    CHECK_CUDA_ERROR(hipEventRecord(m_impl->start_event));

    while (nchange && step <= maxstep) {
        // Reset change counter
        CHECK_CUDA_ERROR(hipMemset(m_impl->d_changes, 0, sizeof(int)));
        
        // Launch kernel
        percolate_kernel<<<gridDim, blockDim, smem_size>>>
            (M, N, d_current, d_next, m_impl->d_changes);
        CHECK_CUDA_ERROR(hipGetLastError());
        
        // Get number of changes
        CHECK_CUDA_ERROR(hipMemcpy(m_impl->h_changes, m_impl->d_changes,
                                   sizeof(int), hipMemcpyDeviceToHost));
        CHECK_CUDA_ERROR(hipDeviceSynchronize());
        nchange = *(m_impl->h_changes);

        if (step % printfreq == 0) {
            std::printf("percolate: number of changes on step %d is %d\n",
                       step, nchange);
        }

        std::swap(d_next, d_current);
        step++;
    }
    
    // Record kernel stop time
    CHECK_CUDA_ERROR(hipEventRecord(m_impl->stop_event));
    CHECK_CUDA_ERROR(hipEventSynchronize(m_impl->stop_event));
    CHECK_CUDA_ERROR(hipEventElapsedTime(&m_impl->kernel_ms,
                                         m_impl->start_event,
                                         m_impl->stop_event));

    // Ensure final state is in d_state
    if (d_current != m_impl->d_state) {
        CHECK_CUDA_ERROR(hipMemcpy(m_impl->d_state, d_current,
                                   m_impl->size() * sizeof(int),
                                   hipMemcpyDeviceToDevice));
    }
    
    // Print timing information
    printf("\nPerformance Summary:\n");
    printf("Kernel execution time: %.3f ms\n", m_impl->kernel_ms);
    printf("Memory transfer time: %.3f ms\n", m_impl->transfer_ms);
    printf("Total time: %.3f ms\n", m_impl->kernel_ms + m_impl->transfer_ms);
}
