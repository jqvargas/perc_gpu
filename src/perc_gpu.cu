#include "hip/hip_runtime.h"
// -*- mode: C++; -*-
//
// Copyright (C) 2025, Rupert Nash, The University of Edinburgh.
//
// All rights reserved.
//
// This file is provided to you to complete an assessment and for
// subsequent private study. It may not be shared and, in particular,
// may not be posted on the internet. Sharing this or any modified
// version may constitute academic misconduct under the University's
// regulations.

#include "perc_gpu.h"

#include <cstdio>
#include <cstring>
#include <vector>
#include <hip/hip_runtime.h>

// Configuration constants
constexpr int BLOCK_SIZE = 32;  // Increased from 16 to 32 for better occupancy
constexpr int printfreq = 100;
constexpr int SHARED_MEM_PADDING = 1;  // Padding to avoid bank conflicts

// CUDA error checking macro
#define CHECK_CUDA_ERROR(val) check_cuda( (val), #val, __FILE__, __LINE__ )
template<typename T>
void check_cuda(T result, char const *const func, const char *const file, int const line) {
    if (result) {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \n",
                file, line, static_cast<unsigned int>(result), func);
        exit(EXIT_FAILURE);
    }
}

// Optimized CUDA kernel using shared memory and improved memory access patterns
__global__ void percolate_kernel(int M, int N, const int* __restrict__ state, 
                                int* __restrict__ next, int* changes) {
    extern __shared__ int shared_state[];
    
    // Calculate global and local indices
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int bx = blockIdx.x * (blockDim.x - 2);  // Overlap blocks by 1 cell
    const int by = blockIdx.y * (blockDim.y - 2);
    const int i = by + ty;
    const int j = bx + tx;
    
    // Shared memory dimensions including halo
    const int smem_pitch = BLOCK_SIZE + SHARED_MEM_PADDING;
    const int smem_idx = ty * smem_pitch + tx;
    
    // Load data into shared memory including halo regions
    if (i <= M && j <= N) {
        const int global_idx = i * (N + 2) + j;
        shared_state[smem_idx] = state[global_idx];
        
        // Load halo regions if thread is on block boundary
        if (tx == 0 && j > 0) {
            shared_state[smem_idx - 1] = state[global_idx - 1];
        }
        if (tx == blockDim.x - 1 && j < N) {
            shared_state[smem_idx + 1] = state[global_idx + 1];
        }
        if (ty == 0 && i > 0) {
            shared_state[smem_idx - smem_pitch] = state[global_idx - (N + 2)];
        }
        if (ty == blockDim.y - 1 && i < M) {
            shared_state[smem_idx + smem_pitch] = state[global_idx + (N + 2)];
        }
    }
    
    __syncthreads();
    
    // Process only interior points
    if (i >= 1 && i <= M && j >= 1 && j <= N) {
        const int oldval = shared_state[smem_idx];
        int newval = oldval;
        
        if (oldval != 0) {
            // Use shared memory for neighbor access
            newval = max(newval, shared_state[smem_idx - smem_pitch]);  // Up
            newval = max(newval, shared_state[smem_idx + smem_pitch]);  // Down
            newval = max(newval, shared_state[smem_idx - 1]);          // Left
            newval = max(newval, shared_state[smem_idx + 1]);          // Right
            
            // Write result to global memory
            next[i * (N + 2) + j] = newval;
            
            // Use warp-level primitives to reduce atomic operations
            unsigned mask = __ballot_sync(0xffffffff, newval != oldval);
            if (mask) {
                if (threadIdx.x == 0) {
                    atomicAdd(changes, __popc(mask));
                }
            }
        } else {
            next[i * (N + 2) + j] = 0;
        }
    }
}

struct GpuRunner::Impl {
    int M;
    int N;
    int* state;      // Host memory (pinned)
    int* tmp;        // Host memory (pinned)
    int* d_state;    // Device memory
    int* d_tmp;      // Device memory
    int* d_changes;  // Device memory for counting changes
    int* h_changes;  // Host memory for changes (pinned)
    
    // CUDA streams for async operations
    hipStream_t compute_stream;
    hipStream_t transfer_stream;
    
    // CUDA events for timing
    hipEvent_t start_event;
    hipEvent_t stop_event;
    hipEvent_t transfer_start;
    hipEvent_t transfer_stop;
    
    float kernel_ms;    // Kernel execution time
    float transfer_ms;  // Transfer time

    Impl(int m, int n) : M(m), N(n), kernel_ms(0), transfer_ms(0) {
        // Allocate pinned memory for better transfer speeds
        CHECK_CUDA_ERROR(hipHostAlloc(&state, size() * sizeof(int), hipHostMallocDefault));
        CHECK_CUDA_ERROR(hipHostAlloc(&tmp, size() * sizeof(int), hipHostMallocDefault));
        CHECK_CUDA_ERROR(hipHostAlloc(&h_changes, sizeof(int), hipHostMallocDefault));
        
        // Allocate device memory
        CHECK_CUDA_ERROR(hipMalloc(&d_state, size() * sizeof(int)));
        CHECK_CUDA_ERROR(hipMalloc(&d_tmp, size() * sizeof(int)));
        CHECK_CUDA_ERROR(hipMalloc(&d_changes, sizeof(int)));
        
        // Create CUDA streams
        CHECK_CUDA_ERROR(hipStreamCreate(&compute_stream));
        CHECK_CUDA_ERROR(hipStreamCreate(&transfer_stream));
        
        // Create timing events
        CHECK_CUDA_ERROR(hipEventCreate(&start_event));
        CHECK_CUDA_ERROR(hipEventCreate(&stop_event));
        CHECK_CUDA_ERROR(hipEventCreate(&transfer_start));
        CHECK_CUDA_ERROR(hipEventCreate(&transfer_stop));
    }
    
    ~Impl() {
        // Free pinned memory
        CHECK_CUDA_ERROR(hipHostFree(state));
        CHECK_CUDA_ERROR(hipHostFree(tmp));
        CHECK_CUDA_ERROR(hipHostFree(h_changes));
        
        // Free device memory
        hipFree(d_state);
        hipFree(d_tmp);
        hipFree(d_changes);
        
        // Destroy streams and events
        hipStreamDestroy(compute_stream);
        hipStreamDestroy(transfer_stream);
        hipEventDestroy(start_event);
        hipEventDestroy(stop_event);
        hipEventDestroy(transfer_start);
        hipEventDestroy(transfer_stop);
    }

    int size() const {
        return (M + 2) * (N + 2);
    }
};

GpuRunner::GpuRunner(int M, int N) : m_impl(std::make_unique<Impl>(M, N)) {
}

GpuRunner::~GpuRunner() = default;

void GpuRunner::copy_in(int const* source) {
    // Record transfer start time
    CHECK_CUDA_ERROR(hipEventRecord(m_impl->transfer_start, m_impl->transfer_stream));
    
    // Copy to pinned memory then to device asynchronously
    std::copy(source, source + m_impl->size(), m_impl->state);
    CHECK_CUDA_ERROR(hipMemcpyAsync(m_impl->d_state, m_impl->state,
                                    m_impl->size() * sizeof(int),
                                    hipMemcpyHostToDevice,
                                    m_impl->transfer_stream));
    
    // Record transfer end time
    CHECK_CUDA_ERROR(hipEventRecord(m_impl->transfer_stop, m_impl->transfer_stream));
    CHECK_CUDA_ERROR(hipEventSynchronize(m_impl->transfer_stop));
    CHECK_CUDA_ERROR(hipEventElapsedTime(&m_impl->transfer_ms,
                                         m_impl->transfer_start,
                                         m_impl->transfer_stop));
}

void GpuRunner::copy_out(int* dest) const {
    // Record transfer start time
    CHECK_CUDA_ERROR(hipEventRecord(m_impl->transfer_start, m_impl->transfer_stream));
    
    // Copy from device to pinned memory then to destination
    CHECK_CUDA_ERROR(hipMemcpyAsync(m_impl->state, m_impl->d_state,
                                    m_impl->size() * sizeof(int),
                                    hipMemcpyDeviceToHost,
                                    m_impl->transfer_stream));
    CHECK_CUDA_ERROR(hipStreamSynchronize(m_impl->transfer_stream));
    std::copy(m_impl->state, m_impl->state + m_impl->size(), dest);
    
    // Record transfer end time
    CHECK_CUDA_ERROR(hipEventRecord(m_impl->transfer_stop, m_impl->transfer_stream));
    CHECK_CUDA_ERROR(hipEventSynchronize(m_impl->transfer_stop));
    float transfer_ms;
    CHECK_CUDA_ERROR(hipEventElapsedTime(&transfer_ms,
                                         m_impl->transfer_start,
                                         m_impl->transfer_stop));
    m_impl->transfer_ms += transfer_ms;
}

void GpuRunner::run() {
    int const M = m_impl->M;
    int const N = m_impl->N;
    
    // Calculate optimal grid and block dimensions
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((N + BLOCK_SIZE - 3) / (BLOCK_SIZE - 2),
                 (M + BLOCK_SIZE - 3) / (BLOCK_SIZE - 2));
    
    // Calculate shared memory size with padding
    size_t smem_size = (BLOCK_SIZE + SHARED_MEM_PADDING) * 
                       (BLOCK_SIZE + SHARED_MEM_PADDING) * sizeof(int);

    int const maxstep = 4 * std::max(M, N);
    int step = 1;
    int nchange = 1;

    // Use pointers to device buffers
    int* d_current = m_impl->d_state;
    int* d_next = m_impl->d_tmp;
    
    // Record kernel start time
    CHECK_CUDA_ERROR(hipEventRecord(m_impl->start_event, m_impl->compute_stream));

    while (nchange && step <= maxstep) {
        // Reset change counter
        CHECK_CUDA_ERROR(hipMemsetAsync(m_impl->d_changes, 0, sizeof(int),
                                        m_impl->compute_stream));
        
        // Launch kernel with shared memory
        percolate_kernel<<<gridDim, blockDim, smem_size, m_impl->compute_stream>>>
            (M, N, d_current, d_next, m_impl->d_changes);
        CHECK_CUDA_ERROR(hipGetLastError());
        
        // Get number of changes asynchronously
        CHECK_CUDA_ERROR(hipMemcpyAsync(m_impl->h_changes, m_impl->d_changes,
                                        sizeof(int), hipMemcpyDeviceToHost,
                                        m_impl->compute_stream));
        CHECK_CUDA_ERROR(hipStreamSynchronize(m_impl->compute_stream));
        nchange = *(m_impl->h_changes);

        if (step % printfreq == 0) {
            std::printf("percolate: number of changes on step %d is %d\n",
                       step, nchange);
        }

        std::swap(d_next, d_current);
        step++;
    }
    
    // Record kernel stop time
    CHECK_CUDA_ERROR(hipEventRecord(m_impl->stop_event, m_impl->compute_stream));
    CHECK_CUDA_ERROR(hipEventSynchronize(m_impl->stop_event));
    CHECK_CUDA_ERROR(hipEventElapsedTime(&m_impl->kernel_ms,
                                         m_impl->start_event,
                                         m_impl->stop_event));

    // Ensure final state is in d_state
    if (d_current != m_impl->d_state) {
        CHECK_CUDA_ERROR(hipMemcpyAsync(m_impl->d_state, d_current,
                                        m_impl->size() * sizeof(int),
                                        hipMemcpyDeviceToDevice,
                                        m_impl->compute_stream));
    }
    
    // Print timing information
    printf("\nPerformance Summary:\n");
    printf("Kernel execution time: %.3f ms\n", m_impl->kernel_ms);
    printf("Memory transfer time: %.3f ms\n", m_impl->transfer_ms);
    printf("Total time: %.3f ms\n", m_impl->kernel_ms + m_impl->transfer_ms);
}
